#include <iostream>
#include <hip/hip_runtime.h>

#define NUM_PRODUCTS 10000

// Kernel 1: Apply a moving average filter to distance data
__global__ void filter_distance(float *distances, float *filtered_distances, int N, int window_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        float sum = 0.0f;
        int count = 0;
        for (int i = idx - window_size; i <= idx + window_size; i++) {
            if (i >= 0 && i < N) {
                sum += distances[i];
                count++;
            }
        }
        filtered_distances[idx] = sum / count;
    }
}

// Kernel 2: Apply a moving average filter to weight data
__global__ void filter_weight(float *weights, float *filtered_weights, int N, int window_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        float sum = 0.0f;
        int count = 0;
        for (int i = idx - window_size; i <= idx + window_size; i++) {
            if (i >= 0 && i < N) {
                sum += weights[i];
                count++;
            }
        }
        filtered_weights[idx] = sum / count;
    }
}

// Kernel 3: Check distance thresholds
__global__ void check_distance_threshold(float *filtered_distances, int *distance_status, float min_dist, float max_dist, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        if (filtered_distances[idx] >= min_dist && filtered_distances[idx] <= max_dist) {
            distance_status[idx] = 1;  // Accepted
        } else {
            distance_status[idx] = 0;  // Rejected
        }
    }
}

// Kernel 4: Check weight thresholds
__global__ void check_weight_threshold(float *filtered_weights, int *weight_status, float min_weight, float max_weight, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        if (filtered_weights[idx] >= min_weight && filtered_weights[idx] <= max_weight) {
            weight_status[idx] = 1;  // Accepted
        } else {
            weight_status[idx] = 0;  // Rejected
        }
    }
}

// Kernel 5: Update product status based on distance and weight checks
__global__ void update_product_status(int *distance_status, int *weight_status, int *product_status, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        if (distance_status[idx] == 1 && weight_status[idx] == 1) {
            product_status[idx] = 1;  // Accepted
        } else {
            product_status[idx] = 0;  // Rejected
        }
    }
}

// Host code
int main() {
    const int size = NUM_PRODUCTS * sizeof(float);
    const int status_size = NUM_PRODUCTS * sizeof(int);

    // Allocate host memory
    float *h_distances = new float[NUM_PRODUCTS];
    float *h_weights = new float[NUM_PRODUCTS];
    float *h_filtered_distances = new float[NUM_PRODUCTS];
    float *h_filtered_weights = new float[NUM_PRODUCTS];
    int *h_distance_status = new int[NUM_PRODUCTS];
    int *h_weight_status = new int[NUM_PRODUCTS];
    int *h_product_status = new int[NUM_PRODUCTS];

    // Initialize random distance and weight data
    for (int i = 0; i < NUM_PRODUCTS; i++) {
        h_distances[i] = static_cast<float>(rand() % 100 + 50);  // Distance in cm
        h_weights[i] = static_cast<float>(rand() % 50 + 10);     // Weight in grams
    }

    // Allocate device memory
    float *d_distances, *d_weights, *d_filtered_distances, *d_filtered_weights;
    int *d_distance_status, *d_weight_status, *d_product_status;
    hipMalloc((void **)&d_distances, size);
    hipMalloc((void **)&d_weights, size);
    hipMalloc((void **)&d_filtered_distances, size);
    hipMalloc((void **)&d_filtered_weights, size);
    hipMalloc((void **)&d_distance_status, status_size);
    hipMalloc((void **)&d_weight_status, status_size);
    hipMalloc((void **)&d_product_status, status_size);

    // Copy data from host to device
    hipMemcpy(d_distances, h_distances, size, hipMemcpyHostToDevice);
    hipMemcpy(d_weights, h_weights, size, hipMemcpyHostToDevice);

    // Configure thread and block dimensions
    int blockSize = 256;
    int numBlocks = (NUM_PRODUCTS + blockSize - 1) / blockSize;

    // Launch kernel routines
    filter_distance<<<numBlocks, blockSize>>>(d_distances, d_filtered_distances, NUM_PRODUCTS, 3);
    filter_weight<<<numBlocks, blockSize>>>(d_weights, d_filtered_weights, NUM_PRODUCTS, 3);
    check_distance_threshold<<<numBlocks, blockSize>>>(d_filtered_distances, d_distance_status, 60.0f, 120.0f, NUM_PRODUCTS);
    check_weight_threshold<<<numBlocks, blockSize>>>(d_filtered_weights, d_weight_status, 15.0f, 40.0f, NUM_PRODUCTS);
    update_product_status<<<numBlocks, blockSize>>>(d_distance_status, d_weight_status, d_product_status, NUM_PRODUCTS);

    // Copy results back to host
    hipMemcpy(h_filtered_distances, d_filtered_distances, size, hipMemcpyDeviceToHost);
    hipMemcpy(h_filtered_weights, d_filtered_weights, size, hipMemcpyDeviceToHost);
    hipMemcpy(h_product_status, d_product_status, status_size, hipMemcpyDeviceToHost);

    // Print results
    for (int i = 0; i < 10; i++) {
        std::cout << "Product " << i << ": Distance = " << h_filtered_distances[i]
                  << " cm, Weight = " << h_filtered_weights[i]
                  << " g, Status = " << (h_product_status[i] ? "Accepted" : "Rejected") << std::endl;
    }

    // Free memory
    delete[] h_distances;
    delete[] h_weights;
    delete[] h_filtered_distances;
    delete[] h_filtered_weights;
    delete[] h_distance_status;
    delete[] h_weight_status;
    delete[] h_product_status;
    hipFree(d_distances);
    hipFree(d_weights);
    hipFree(d_filtered_distances);
    hipFree(d_filtered_weights);
    hipFree(d_distance_status);
    hipFree(d_weight_status);
    hipFree(d_product_status);

    return 0;
}

